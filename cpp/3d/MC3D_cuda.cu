#include <stdio.h> // for fprintf, stderr
#include "hiprand/hiprand_kernel.h"

#include "MC3D_cuda.hpp"
#include "MC3D_kernels.cuh"

#define gpuErrchk(ans) { gpuAssert((ans), __FILE__, __LINE__); }
inline void gpuAssert(hipError_t code, const char *file, int line, bool abort=true) {
   if (code != hipSuccess) {
    fprintf(stderr,"GPUassert: %s %s %d\n", hipGetErrorString(code), file, line);
    if (abort) exit(code);
   }
}


namespace util {
  int invoke_ray_triangle_intersects (
    double O[3],
    double D[3],
    double V0[3],
    double V1[3],
    double V2[3],
    double *t
  )
  {
    return util::ray_triangle_intersects(O, D, V0, V1, V2, t);
  }

  void invoke_cross (double dest[3], double v1[3], double v2[3])
  {
    return util::cross (dest, v1, v2);
  }

  double invoke_dot (double v1[3], double v2[3])
  {
    return util::dot (v1, v2);
  }

  void invoke_sub (double dest[3], double v1[3], double v2[3])
  {
    return util::sub (dest, v1, v2);
  }

  void invoke_init_random (
    unsigned long long seed, unsigned long long sequence,
    unsigned long long offset, hiprandState_t state
  )
  {
    util::init_random<<<1, 1>>>(seed, sequence, offset, state);
    gpuErrchk(hipGetLastError());
    gpuErrchk(hipDeviceSynchronize());
  }

  void invoke_uniform_closed (
    hiprandState_t state, std::vector<double>& res
  )
  {
    double* device_res;

    gpuErrchk(
      hipMalloc((void**) &device_res, res.size()*sizeof(double)));

    util::random<double><<<1, 1>>>(
      state, device_res, res.size());
    gpuErrchk(hipGetLastError());
    gpuErrchk(hipDeviceSynchronize());

    gpuErrchk(
      hipMemcpy(res.data(), device_res,
        res.size()*sizeof(double), hipMemcpyDeviceToHost));
    gpuErrchk(
      hipFree(device_res));
  }

  // template<typename StateType>
  // double invoke_uniform_open (StateType* state)
  // {
  //   return util::uniform_open (state);
  // }
  // template double invoke_uniform_open (hiprandState_t* state) ;
  //
  // template<typename StateType>
  // double invoke_uniform_half_upper (StateType* state)
  // {
  //   return util::uniform_half_upper (state);
  // }
  // template double invoke_uniform_half_upper (hiprandState_t* state) ;
  //
  // template<typename StateType>
  // double invoke_uniform_half_lower (StateType* state)
  // {
  //   return util::uniform_half_lower (state);
  // }
  // template double invoke_uniform_half_lower (hiprandState_t* state) ;

}
