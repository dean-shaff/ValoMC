#include "hip/hip_runtime.h"
#include <string>

#include "catch.hpp"

#include "MC3D_cuda.cuh"
#include "util.hpp"

static ValoMC::test::util::TestConfig config;

TEST_CASE(
  "MC3DCUDA can calculate the amount of space needed ",
  "[MC3DCUDA][unit][get_total_memory_usage]"
)
{

  unsigned states_size = 100;

  config.init_MC3D_from_json();

  ValoMC::MC3DCUDA mc3dcuda(config.get_mc3d(), 0);

  unsigned int baseline = mc3dcuda.get_total_memory_usage();
  mc3dcuda.set_states_size(states_size);
  unsigned int usage = mc3dcuda.get_total_memory_usage();

  REQUIRE(usage - baseline == sizeof(hiprandState_t)*states_size);
}

TEST_CASE(
  "MC3DCUDA can initialize properties from MC3D object",
  "[MC3DCUDA][unit][init]"
)
{
  config.init_MC3D_from_json();
  MC3D mc3d = config.get_mc3d();

  mc3d.Nphoton = 100;

  ValoMC::MC3DCUDA mc3dcuda(mc3d, 1);

  mc3dcuda.init();

  CHECK(mc3dcuda.get_nphotons() == 100);
}

__global__ void iter_states(
  hiprandState_t* states,
  unsigned states_size,
  unsigned* result
) {
  const unsigned idx = threadIdx.x + blockDim.x * blockIdx.x;
  if (idx == 0) {
    for (unsigned istate=0; istate<states_size; istate++) {
      states[istate];
      (*result)++;
    }
  }
}


TEST_CASE(
  "MC3DCUDA can allocate memory",
  "[MC3DCUDA][unit][allocate]"
)
{
  unsigned states_size = 10;
  config.init_MC3D_from_json();

  ValoMC::MC3DCUDA mc3dcuda(config.get_mc3d(), states_size);

  mc3dcuda.allocate();
  unsigned result_h = 0;
  unsigned* result_d;
  gpuErrchk(hipMalloc((void**)&result_d, sizeof(unsigned)));
  gpuErrchk(hipMemcpy(result_d, &result_h, sizeof(unsigned), hipMemcpyHostToDevice));
  iter_states<<<1,1>>>(mc3dcuda.get_states(), mc3dcuda.get_states_size(), result_d);
  gpuErrchk(hipGetLastError());
  gpuErrchk(hipMemcpy(&result_h, result_d, sizeof(unsigned), hipMemcpyDeviceToHost));

  REQUIRE(result_h == mc3dcuda.get_states_size());

  gpuErrchk(hipDeviceSynchronize());
}

template<typename T>
__global__ void iter_boundary (Array<T>* boundary, unsigned* result)
{
  const unsigned idx = threadIdx.x + blockDim.x * blockIdx.x;
  if (idx == 0) {
    for (unsigned ib=0; ib<boundary->N; ib++) {
      (*boundary)[ib];
      (*result)++;
    }
  }
}


TEST_CASE(
  "MC3DCUDA can transfer arrays from MC3D object to internal Array objects",
  "[MC3DCUDA][unit][h2d]"
)
{
  config.init_MC3D_from_json();

  ValoMC::MC3DCUDA mc3dcuda(config.get_mc3d(), 1);

  mc3dcuda.allocate();
  mc3dcuda.h2d();
  unsigned result_h = 0;
  unsigned* result_d;
  gpuErrchk(hipMalloc((void**)&result_d, sizeof(unsigned)));
  gpuErrchk(hipMemcpy(result_d, &result_h, sizeof(unsigned), hipMemcpyHostToDevice));
  iter_boundary<<<1,1>>>(mc3dcuda.get_boundary(), result_d);
  gpuErrchk(hipGetLastError());
  gpuErrchk(hipMemcpy(&result_h, result_d, sizeof(unsigned), hipMemcpyDeviceToHost));

  REQUIRE(result_h == config.get_mc3d().BH.N);

  gpuErrchk(hipDeviceSynchronize());

}


template<typename T>
__global__ void fill_array (Array<T>* arr, T val)
{
  const unsigned idx = threadIdx.x + blockDim.x * blockIdx.x;
  const unsigned total_size_x = blockDim.x * gridDim.x;

  for (unsigned iarr=idx; iarr<arr->N; iarr+=total_size_x) {
    (*arr)[iarr] = val;
  }
}


TEST_CASE(
  "MC3DCUDA can transfer result arrays to MC3D object",
  "[MC3DCUDA][unit][d2h]"
)
{
  config.init_MC3D_from_json();

  MC3D mc3d = config.get_mc3d();

  ValoMC::MC3DCUDA mc3dcuda(mc3d, 1);

  mc3dcuda.allocate();

  unsigned ER_N = mc3d.ER.N;

  unsigned block_size = 1024;
  unsigned grid_size = 1;
  fill_array<<<block_size, grid_size>>> (mc3dcuda.get_pow_den_vol_real(), 1.0);
  gpuErrchk(hipGetLastError());
  mc3dcuda.d2h();

  bool allclose = true;

  for (unsigned idx=0; idx<ER_N; idx++) {
    if (mc3d.ER[idx] != 1.0) {
      allclose = false;
    }
  }

  REQUIRE(allclose == true);
  gpuErrchk(hipDeviceSynchronize());
}
