#include "hip/hip_runtime.h"
#include <iostream>

#include "catch.hpp"

#include "GPUArray.cuh"

template<typename T>
__global__ void iter_array (GPUArray<T>* arr) {
  const unsigned idx = threadIdx.x + blockIdx.x * blockDim.x;
}


TEMPLATE_TEST_CASE (
  "ensure constructors and copy operators work",
  "[GPUArray][cuda][unit]",
  float
)
{
  SECTION ("Empty Constructor") {
    GPUArray<TestType> arr;
    REQUIRE(arr.data == NULL);
    REQUIRE(arr.IsGPU == 0);
    REQUIRE(arr.IsRef == 0);
    REQUIRE(arr.rank == 0);
  }

  SECTION ("Copy Constructor") {
    GPUArray<TestType> arr;
    GPUArray<TestType> arr1(arr);
    REQUIRE(arr1.IsRef == 1);
  }

  SECTION ("Copy Operator") {
    GPUArray<TestType> arr;
    GPUArray<TestType> arr1 = arr;
    REQUIRE(arr1.IsRef == 1);
    arr.IsGPU = 1;
    REQUIRE_THROWS(arr1 = arr);
  }
}

TEMPLATE_TEST_CASE (
  "Ensure resize works", "[GPUArray][cuda][unit]",
  float
)
{
  SECTION ("resize works on host array") {
    GPUArray<TestType> arrh;
    arrh.resize(10);
    arrh[0] = 1;
  }

  SECTION ("resize works on device array") {
    GPUArray<TestType> arrd;
    arrd.IsGPU = 1;
    arrd.resize(10);
    arrd[0] = 1;
  }
}

TEMPLATE_TEST_CASE (
  "ensure GPUArray h2d works",
  "[GPUArray][cuda][unit][h2d]",
  float
)
{
  GPUArray<TestType> arrh;
  GPUArray<TestType> arrd;
  arrh.resize(10);
  arrh.h2d(arrd);

  arrh.IsGPU = 1;
  REQUIRE_THROWS(arrh.h2d(arrd));
}

TEMPLATE_TEST_CASE (
  "ensure GPUArray d2h works",
  "[GPUArray][cuda][unit][d2h]",
  float
)
{
  GPUArray<TestType> arrd;
  GPUArray<TestType> arrh;
  arrd.IsGPU = 1;
  arrd.resize(10);
  arrd.d2h(arrh);

  arrd.IsGPU = 0;
  REQUIRE_THROWS(arrd.d2h(arrh));
}
