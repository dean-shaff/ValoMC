#include "hip/hip_runtime.h"
#include <random>
#include <vector>
#include <numeric>

#include "catch.hpp"

#include "MC3D_util.cuh"
#include "MC3D.hpp"

TEST_CASE(
  "Vector Ray Triangle Intersects produces same result",
  "[ray_triangle_intersects][RayTriangleIntersects][unit][util][cuda]"
)
{
  const unsigned ntest = 100;
  // long unsigned seed = static_cast<long unsigned unsigned>(
  //   std::chrono::high_resolution_clock::now().time_since_epoch().count());
  // std::default_random_engine generator(seed);

  // std::random_device rd;
  // std::mt19937 generator(rd());

  std::default_random_engine generator;
  std::uniform_real_distribution<double> real (0.0, 1.0);

  std::vector<std::vector<double>> expected (5, std::vector<double>(3, 0.0));
  std::vector<std::vector<double>> test;
  double t_expected;
  double t_test;

  unsigned nsame = 0;

  for (unsigned itest=0; itest<ntest; itest++) {
    for (unsigned idx=0; idx<expected.size(); idx++) {
      for (unsigned idy=0; idy<expected[idx].size(); idy++) {
        expected[idx][idy] = real(generator);
      }
    }
    test = expected;

    unsigned res_expected = RayTriangleIntersects(
      expected[0].data(), expected[1].data(),
      expected[2].data(), expected[3].data(),
      expected[4].data(), &t_expected
    );
    unsigned res_test = ValoMC::util::ray_triangle_intersects(
      test[0].data(), test[1].data(),
      test[2].data(), test[3].data(),
      test[4].data(), &t_test
    );

    if (res_expected == res_test && t_expected == t_test) {
      nsame++;
    }
    t_expected = 0.0;
    t_test = 0.0;
  }
  REQUIRE(nsame == ntest);
}



TEMPLATE_TEST_CASE(
  "check_ptr determines if pointer was allocated on device",
  "[unit][util][cuda][check_ptr]",
  float, Array<float>
)
{
  // TestType* ptr;
  // CHECK(ValoMC::util::check_device_ptr(ptr) == false);

  TestType* ptr_d;
  gpuErrchk(hipMalloc((void**)&ptr_d, sizeof(TestType)));
  CHECK(ValoMC::util::check_device_ptr(ptr_d) == true);
  gpuErrchk(hipFree(ptr_d));

  TestType* ptr_h;
  gpuErrchk(hipHostMalloc((void**)&ptr_h, sizeof(TestType)));
  CHECK(ValoMC::util::check_device_ptr(ptr_h) == false);
  gpuErrchk(hipHostFree(ptr_h));

  gpuErrchk(hipDeviceSynchronize());
}


template<typename T>
__global__ void test_h2d (Array<T>* arr, unsigned* result) {
  const unsigned idx = threadIdx.x + blockIdx.x * blockDim.x;
  if (idx == 0) {
    for (unsigned istep=0; istep<arr->N; istep++) {
      (*arr)[istep];
      *result += 1;
    }
  }
}


TEMPLATE_TEST_CASE(
  "h2d works as expected",
  "[util][unit][cuda][h2d]",
  float, double
)
{
  Array<TestType> arr_h;
  arr_h.resize(10, 2);
  for (unsigned idx=0; idx<arr_h.Nx; idx++) {
    for (unsigned idy=0; idy<arr_h.Ny; idy++) {
      arr_h(idx, idy) = static_cast<TestType>(idx + arr_h.Nx*idy);
    }
  }

  Array<TestType>* arr;
  gpuErrchk(hipMalloc((void**)&arr, sizeof(Array<TestType>)));

  SECTION ("h2d works in isolation") {
    ValoMC::util::h2d(arr, &arr_h);
  }

  SECTION ("can pass Array pointer to CUDA kernel") {
    ValoMC::util::h2d(arr, &arr_h);
    unsigned* result;
    unsigned result_h = 0;
    gpuErrchk(hipMalloc((void**)&result, sizeof(unsigned)));
    test_h2d<TestType><<<1, 1>>>(arr, result);
    gpuErrchk(hipGetLastError());
    gpuErrchk(hipMemcpy(&result_h, result, sizeof(unsigned), hipMemcpyDeviceToHost));
    gpuErrchk(hipFree(result));
    REQUIRE(result_h == 20);
  }
  gpuErrchk(hipFree(arr));
  gpuErrchk(hipDeviceSynchronize());
}


template<typename T>
__global__ void test_d2h (Array<T>* arr, T val) {
  const unsigned idx = threadIdx.x + blockIdx.x * blockDim.x;
  // prunsignedf("test_d2h: val=%f\n", val);
  if (idx > arr->N) {
    return;
  }
  for (unsigned istep=idx; istep<arr->N; istep++) {
    (*arr)[istep] = val;
  }
}

TEMPLATE_TEST_CASE(
  "d2h works as expected",
  "[util][unit][cuda][d2h]",
  float, double
)
{
  TestType val = 1.0;
  Array<TestType> arr_h;
  arr_h.resize(10, 2);

  Array<TestType>* arr;
  gpuErrchk(hipMalloc((void**)&arr, sizeof(Array<TestType>)));
  ValoMC::util::h2d(arr, &arr_h);

  test_d2h<TestType><<<1, 1>>>(arr, val);
  gpuErrchk(hipGetLastError());
  gpuErrchk(hipDeviceSynchronize());
  ValoMC::util::d2h(&arr_h, arr);

  bool all_close = true;

  for (unsigned idx=0; idx<arr_h.N; idx++) {
    if (arr_h[idx] != val) {
      all_close = false;
    }
  }
  REQUIRE(all_close == true);

  gpuErrchk(hipFree(arr));
  gpuErrchk(hipDeviceSynchronize());
}


class ContainsArray {
public:
  ContainsArray () {
    // std::cerr << "ContainsArray::ContainsArray" << std::endl;
    is_allocated = false;
  }

  ~ContainsArray () {
    // std::cerr << "ContainsArray::~ContainsArray" << std::endl;
  }

  void allocate () {
    gpuErrchk(hipMalloc((void**)&arr, sizeof(Array<double>)));
    is_allocated = true;
  }

  void destroy () {
    // std::cerr << "ContainsArray::destroy" << std::endl;
    if (is_allocated) {
      gpuErrchk(hipFree(arr));
      is_allocated = false;
    }
  }

  Array<double>* arr;

protected:

  bool is_allocated;
};

// class PseudoContainsArray {
//   char buffer[sizeof(ContainsArray)];
// };
//
//
// __global__ void test_ContainsArray (PseudoContainsArray _con, unsigned* result)
// {
//   ContainsArray &con = *((ContainsArray *)&_con);
//
//   const unsigned idx = threadIdx.x + blockIdx.x * blockDim.x;
//   Array<double> arr = *(con.arr);
//   if (idx == 0) {
//     for (unsigned istep=0; istep<arr.N; istep++) {
//       arr[istep];
//       *result += 1;
//     }
//   }
// }

__global__ void test_ContainsArray (ContainsArray con, unsigned* result)
{
  const unsigned idx = threadIdx.x + blockIdx.x * blockDim.x;
  Array<double> arr = *(con.arr);
  if (idx == 0) {
    for (unsigned istep=0; istep<arr.N; istep++) {
      arr[istep];
      *result += 1;
    }
  }
}


TEST_CASE (
  "Can call kernel on object containing pointers to Arrays",
  "[util][cuda][ContainsArray]"
)
{
  ContainsArray con;
  Array<double> arr_h;
  arr_h.resize(10, 2);
  for (unsigned idx=0; idx<arr_h.N; idx++) {
    arr_h[idx] = 1.0;
  }
  con.allocate();
  ValoMC::util::h2d(con.arr, &arr_h);

  unsigned* result;
  unsigned result_h = 0.0;
  gpuErrchk(hipMalloc((void**)&result, sizeof(unsigned)));
  // test_ContainsArray<<<1, 1>>>(*(PseudoContainsArray* )&con, result);
  test_ContainsArray<<<1, 1>>>(con, result);
  gpuErrchk(hipGetLastError());
  gpuErrchk(hipMemcpy(&result_h, result, sizeof(unsigned), hipMemcpyDeviceToHost));
  gpuErrchk(hipFree(result));
  gpuErrchk(hipDeviceSynchronize());
  REQUIRE(result_h == 20);
  con.destroy();
}

class ContainsArrayLarge : public ContainsArray {
public:
  char buffer[4096];
};


__global__ void test_pointer_to_ContainsArray (
  ContainsArrayLarge* con, unsigned* result, double val
)
{
  const unsigned idx = threadIdx.x + blockIdx.x * blockDim.x;
  Array<double> arr = *(con->arr);
  if (idx == 0) {
    for (unsigned istep=0; istep<arr.N; istep++) {
      arr[istep] = val;
      *result += 1;
    }
  }
}


TEST_CASE (
  "Can call kernel on pointer to object containing pointers to Arrays",
  "[util][cuda][ContainsArray]"
)
{
  double val = 1.0;
  ContainsArrayLarge con_h;
  Array<double> arr_h;
  arr_h.resize(10, 2);
  for (unsigned idx=0; idx<arr_h.N; idx++) {
    arr_h[idx] = 0.0;
  }
  con_h.allocate();
  ValoMC::util::h2d(con_h.arr, &arr_h);

  // now create device side:
  ContainsArrayLarge* con_d;
  hipMalloc((void**)&con_d, sizeof(ContainsArrayLarge));
  hipMemcpy(con_d, &con_h, sizeof(ContainsArrayLarge), hipMemcpyHostToDevice);

  unsigned* result;
  unsigned result_h = 0.0;
  gpuErrchk(hipMalloc((void**)&result, sizeof(unsigned)));
  test_pointer_to_ContainsArray<<<1, 1>>>(con_d, result, val);
  gpuErrchk(hipGetLastError());
  gpuErrchk(hipMemcpy(&result_h, result, sizeof(unsigned), hipMemcpyDeviceToHost));
  CHECK(result_h == 20);

  // gpuErrchk(hipMemcpy(&con_h, con_d, sizeof(ContainsArray), hipMemcpyDeviceToHost));

  ValoMC::util::d2h(&arr_h, con_h.arr);
  bool allclose = true;

  for (unsigned idx=0; idx<arr_h.N; idx++) {
    if (arr_h[idx] != val) {
      allclose = false;
    }
  }
  CHECK(allclose == true);

  gpuErrchk(hipFree(result));
  gpuErrchk(hipFree(con_d))
  con_h.destroy();
  gpuErrchk(hipDeviceSynchronize());
}


template<typename T>
__global__ void test_copy_attributes (Array<T>* arr, unsigned* result)
{
  const unsigned idx = threadIdx.x + blockIdx.x * blockDim.x;
  if (idx == 0) {
    result[0] = arr->Nx;
    result[1] = arr->Ny;
    result[2] = arr->Nz;
  }
}


TEMPLATE_TEST_CASE(
  "copy_attributes works as expected",
  "[util][unit][cuda][copy_attributes]",
  float, double
)
{
  Array<TestType> arr_h;
  arr_h.resize(10, 2, 2);

  Array<TestType>* arr;
  gpuErrchk(hipMalloc((void**)&arr, sizeof(Array<TestType>)));
  ValoMC::util::copy_attributes(arr, &arr_h);

  std::vector<unsigned> result_h(3);

  unsigned* result;
  gpuErrchk(hipMalloc((void**)&result, sizeof(unsigned)*3));
  test_copy_attributes<TestType><<<1, 1>>>(arr, result);
  gpuErrchk(hipGetLastError());
  gpuErrchk(hipMemcpy(result_h.data(), result, sizeof(unsigned)*3, hipMemcpyDeviceToHost));
  gpuErrchk(hipFree(result));

  CHECK(result_h[0] == 10);
  CHECK(result_h[1] == 2);
  CHECK(result_h[2] == 2);

  gpuErrchk(hipFree(arr));
  gpuErrchk(hipDeviceSynchronize());
}

template<typename T>
__global__ void test_reserve (Array<T>* arr, unsigned size, unsigned* result)
{
  const unsigned idx = threadIdx.x + blockDim.x * blockIdx.x;
  if (idx == 0) {
    for (unsigned idat=0; idat<size; idat++) {
      // printf("test_reserve idat=%u, result=%u\n", idat, *result);
      arr->data[idat];
      (*result) ++;
    }
  }
}


TEMPLATE_TEST_CASE(
  "reserve works as expected",
  "[util][unit][cuda][reserve]",
  float, double
)
{
  unsigned size = 10;
  Array<TestType>* arr;
  gpuErrchk(hipMalloc((void**)&arr, sizeof(Array<TestType>)));
  ValoMC::util::reserve(arr, size);

  unsigned result_h = 0;
  unsigned* result;
  gpuErrchk(hipMalloc((void**)&result, sizeof(unsigned)));
  gpuErrchk(hipMemcpy(result, &result_h, sizeof(unsigned), hipMemcpyHostToDevice));
  test_reserve<TestType><<<1, 1>>>(arr, size, result);
  gpuErrchk(hipGetLastError());
  gpuErrchk(hipMemcpy(&result_h, result, sizeof(unsigned), hipMemcpyDeviceToHost));
  gpuErrchk(hipFree(result));

  CHECK(result_h == size);

  gpuErrchk(hipFree(arr));
  gpuErrchk(hipDeviceSynchronize());
}


// TEST_CASE("Random number generators produce same statistics", "[unit][util][cuda][random]")
// {
//   unsigned nsamples = 100;
//   hiprandState_t state;
//   util::invoke_init_random(1024, 1, 0, state);
//   std::vector<double> cuda_samples(nsamples);
//
//   SECTION("Uniform Closed") {
//     util::invoke_uniform_closed(state, cuda_samples);
//     double mean = std::accumulate(
//       cuda_samples.begin(), cuda_samples.end(), 0.0) / (double) nsamples;
//     std::cerr << "mean=" << mean << std::endl;
//     // for (unsigned idx=0; idx<cuda_samples.size(); idx++) {
//     //   std::cerr << cuda_samples[idx] << " ";
//     // }
//     // std::cerr << std::endl;
//   }
//
//   // SECTION("Uniform Open") {
//   //
//   // }
//   //
//   // SECTION("Uniform Half Upper") {
//   //
//   // }
//   //
//   // SECTION("Uniform Half Lower") {
//   //
//   // }
// }
